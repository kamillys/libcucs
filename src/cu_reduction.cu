#include "hip/hip_runtime.h"
#include <iostream>

#include "cs_internal.h"
#include "cudalib.h"

#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <bitset>


namespace cu
{

__constant__ static float squad[10*HASHBITS];
static texture<float4, 1, hipReadModeElementType> spinTex;
static texture<hashtype, 1, hipReadModeElementType> coordPartTex;

__device__
inline static int getSign(float v0, float v1, float v2, float v3, int i)
{
    i = i * 10;
    const float s0 = squad[i+0];
    const float s1 = squad[i+1];
    const float s2 = squad[i+2];
    const float s3 = squad[i+3];
    const float s4 = squad[i+4];
    const float s5 = squad[i+5];
    const float s6 = squad[i+6];
    const float s7 = squad[i+7];
    const float s8 = squad[i+8];
    const float s9 = squad[i+9];

    const float sum =
            (s0*v0+s1*v1+s2*v2+s3*v3) * v0 +
            (s1*v0+s4*v1+s5*v2+s6*v3) * v1 +
            (s2*v0+s5*v1+s7*v2+s8*v3) * v2 +
            (s3*v0+s6*v1+s8*v2+s9*v3) * v3;


    return (signbit(sum)) ? 0 : 1;
    //return signbit(sum);
}

struct ComputeHash
{
    u_int32_t count;

    ComputeHash(size_t s)
        : count(s) { }

    __device__
    hashtype operator()(const u_int32_t& index)
    {
        const float v0=tex1Dfetch(spinTex, index).x;
        const float v1=tex1Dfetch(spinTex, index).y;
        const float v2=tex1Dfetch(spinTex, index).z;
        const float v3=tex1Dfetch(spinTex, index).w;

        hashtype retval(0); int offset = 0;
        //for (size_t i=0;i<count;++i)
        for (u_int32_t i=0;i<count;++i)
        {
            retval ^= getSign(v0,v1,v2,v3, i) << offset;
            offset = (offset+1)& HASH_MASK;
        }
        return retval;
    }
};

struct ReorderSpins
{
    __device__
    float4 operator()(const u_int32_t& _a)
    {
        return tex1Dfetch(spinTex, _a);
    }
};

struct SpinHashChecker
{
    __device__
    u_int32_t operator()(const u_int32_t& index, const u_int32_t& value)
    {
        if (index == 0)
            return 0;
        u_int32_t v0 = tex1Dfetch(coordPartTex, index-1);
        u_int32_t v1 = tex1Dfetch(coordPartTex, index);
        if (v0 == v1)
            return value+1;
        return value;
    }
};

struct SpinIndexCleaner
{
    u_int32_t count;

    SpinIndexCleaner(size_t s)
        : count(s) { }

    __device__
    bool operator()(const u_int32_t& index)
    {
        u_int32_t value = tex1Dfetch(coordPartTex, index);
        return value == count;
    }
};

void compute_hash_part(thrust::device_vector<float4>& spins,
                       const std::vector<float>& spinquadrics,
                       thrust::device_vector<hashtype>& hashPart,
                       int i,
                       size_t& rem)
{
    size_t toCpy = std::min<size_t>(rem, HASHBITS);

    cutilSafeCall( hipBindTexture(NULL, spinTex, thrust::raw_pointer_cast(spins.data()), sizeof(float4)*spins.size()) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(squad), thrust::raw_pointer_cast(spinquadrics.data())+i*10*HASHBITS, sizeof(float)*10*toCpy) );

    thrust::transform(thrust::counting_iterator<u_int32_t>(0),
                      thrust::counting_iterator<u_int32_t>(spins.size()),
                      hashPart.begin(), ComputeHash(toCpy));

    cutilSafeCall( hipUnbindTexture(spinTex) );
}

void make_unique_spins(thrust::device_vector<float4>& spins,
                       const std::vector<float>& spinquadrics,
                       size_t spinquadCount)
{
    size_t parts = inc_div<size_t>(spinquadrics.size()/10, HASHBITS);

    /*
     * - Sort & compute parts
     * - Reduce & compute parts
    */

    thrust::device_vector<hashtype> hashPart(spins.size());
    {
        size_t partsSizes;
        size_t rem = spinquadCount;
        for (int i=0;i<parts; ++i)
        {
            partsSizes = std::min<size_t>(rem, HASHBITS);
            rem -= HASHBITS;
            compute_hash_part(spins, spinquadrics, hashPart, i, partsSizes);
            thrust::stable_sort_by_key(hashPart.begin(), hashPart.end(), spins.begin());
        }
    }
    //Make unique
    //Algo: Check if elem before has the same hash
    //if yes, +1
    //Remove all that has counter equal to part count
    {
        thrust::device_vector<u_int32_t> hash_counter(spins.size(), 0);
        thrust::device_vector<u_int32_t> elemsIds(spins.size());
        thrust::sequence(elemsIds.begin(), elemsIds.end());
        size_t partsSizes;
        size_t rem = spinquadCount;
        for (int i=0;i<parts; ++i)
        {
            partsSizes = std::min<size_t>(rem, HASHBITS);
            rem -= HASHBITS;
            compute_hash_part(spins, spinquadrics, hashPart, i, partsSizes);

            cutilSafeCall( hipBindTexture(NULL, coordPartTex, thrust::raw_pointer_cast(hashPart.data()), sizeof(hashtype)*hashPart.size()) );
            thrust::transform(thrust::counting_iterator<u_int32_t>(0),
                              thrust::counting_iterator<u_int32_t>(spins.size()),
                              hash_counter.begin(),
                              hash_counter.begin(),
                              SpinHashChecker());
            cutilSafeCall( hipUnbindTexture(coordPartTex) );
        }
        thrust::sequence(elemsIds.begin(), elemsIds.end());
        cutilSafeCall( hipBindTexture(NULL, coordPartTex, thrust::raw_pointer_cast(hash_counter.data()), sizeof(hashtype)*hash_counter.size()) );
        elemsIds.resize(thrust::remove_if(elemsIds.begin(), elemsIds.end(), SpinIndexCleaner(parts)) - elemsIds.begin());
        cutilSafeCall( hipUnbindTexture(coordPartTex) );

        //std::cerr << "Remaining: " << elemsIds.size() << std::endl;
        //Final Reorder
        thrust::device_vector<float4> final_spins(elemsIds.size());
        cutilSafeCall( hipBindTexture(NULL, spinTex, thrust::raw_pointer_cast(spins.data()), sizeof(float4)*spins.size()) );
        thrust::transform(elemsIds.begin(), elemsIds.end(), final_spins.begin(), ReorderSpins());
        cutilSafeCall( hipUnbindTexture(spinTex) );
        spins.swap(final_spins);
    }
}

}
